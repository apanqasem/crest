#include "hip/hip_runtime.h"
/*
* Copyright 1993-2008 NVIDIA Corporation.  All rights reserved.
*
* NOTICE TO USER:
*
* This source code is subject to NVIDIA ownership rights under U.S. and
* international Copyright laws.
*
* NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
* CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
* IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
* REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
* MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE
* IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
* OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
* OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
* OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE
* OR PERFORMANCE OF THIS SOURCE CODE.
*
* U.S. Government End Users.  This source code is a "commercial item" as
* that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of
* "commercial computer software" and "commercial computer software
* documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995)
* and is provided to the U.S. Government only as a commercial end item.
* Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
* 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
* source code with only those rights set forth herein.
*/

/*
    Bicubic texture filtering sample
    sgreen 6/2008

    This sample demonstrates how to efficiently implement bicubic texture
    filtering in CUDA.

    Bicubic filtering is a higher order interpolation method that produces
    smoother results than bilinear interpolation:
    http://en.wikipedia.org/wiki/Bicubic

    It requires reading a 4 x 4 pixel neighbourhood rather than the
    2 x 2 area required by bilinear filtering.

    Current graphics hardware doesn't support bicubic filtering natively,
    but it is possible to compose a bicubic filter using just 4 bilinear
    lookups by offsetting the sample position within each texel and weighting
    the samples correctly. The only disadvantage to this method is that the
    hardware only maintains 9-bits of filtering precision within each texel.

    See "Fast Third-Order Texture Filtering", Sigg & Hadwiger, GPU Gems 2:
    http://developer.nvidia.com/object/gpu_gems_2_home.html

    v1.1 - updated to include the brute force method using 16 texture lookups.
    
    Example performance results from GeForce 8800 GTS:
    Bilinear     - 5500 MPixels/sec
    Bicubic      - 1400 MPixels/sec
    Fast Bicubic - 2100 MPixels/sec
*/

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <GL/glew.h>

#if defined (__APPLE__) || defined(MACOSX)
#include <GLUT/glut.h>
#else
#include <GL/glut.h>
#endif

#include <cuda_gl_interop.h>
#include <cutil_inline.h>

typedef unsigned int uint;
typedef unsigned char uchar;

#include <bicubicTexture_kernel.cuh>

#define USE_BUFFER_TEX 0

char *imageFilename = "lena_bw.pgm";

uint width = 512, height = 512;
uint imageWidth, imageHeight;
dim3 blockSize(16, 16);
dim3 gridSize(width / blockSize.x, height / blockSize.y);

enum Mode { MODE_NEAREST, MODE_BILINEAR, MODE_BICUBIC, MODE_FAST_BICUBIC };
Mode mode = MODE_FAST_BICUBIC;
bool drawCurves = false;

hipArray *d_imageArray = 0;
GLuint pbo = 0;                 // OpenGL pixel buffer object
GLuint displayTex = 0;
GLuint bufferTex = 0;
GLuint fprog;                   // fragment program (shader)

float tx = 9.0f, ty = 10.0f;    // image translation
float scale = 1.0f / 16.0f;     // image scale
float cx, cy;                   // image centre

bool bQATest = false;

void initPixelBuffer();
void runBenchmark(int iterations);

// render image using CUDA
void render(uchar4 *output)
{
    // call CUDA kernel, writing results to PBO memory
    switch(mode) {
    case MODE_NEAREST:
        tex.filterMode = hipFilterModePoint;
        d_render<<<gridSize, blockSize>>>(output, width, height, tx, ty, scale, cx, cy);
        break;
    case MODE_BILINEAR:
        tex.filterMode = hipFilterModeLinear;
        d_render<<<gridSize, blockSize>>>(output, width, height, tx, ty, scale, cx, cy);
        break;
    case MODE_BICUBIC:
        tex.filterMode = hipFilterModePoint;
        d_renderBicubic<<<gridSize, blockSize>>>(output, width, height, tx, ty, scale, cx, cy);
        break;
    case MODE_FAST_BICUBIC:
        tex.filterMode = hipFilterModeLinear;
        d_renderFastBicubic<<<gridSize, blockSize>>>(output, width, height, tx, ty, scale, cx, cy);
        break;
    }
    cutilCheckMsg("kernel failed");
}

void plotCurve(float (*func)(float))
{
    const int steps = 100;
    glBegin(GL_LINE_STRIP);
    for(int i=0; i<steps; i++) {
        float x = i / (float) (steps-1);
        glVertex2f(x, func(x));
    }
    glEnd();
}

// display results using OpenGL (called by GLUT)
void display()
{
    // map PBO to get CUDA device pointer
    uchar4 *d_output;
    cutilSafeCall(cudaGLMapBufferObject((void**)&d_output, pbo));
    render(d_output);
    cutilSafeCall(cudaGLUnmapBufferObject(pbo));

    // display results
    glClear(GL_COLOR_BUFFER_BIT);

#if USE_BUFFER_TEX
    // display using buffer texture
    glBindTexture(GL_TEXTURE_BUFFER_EXT, bufferTex);
    glBindProgramARB(GL_FRAGMENT_PROGRAM_ARB, fprog);
    glEnable(GL_FRAGMENT_PROGRAM_ARB);
    glProgramLocalParameterI4iNV(GL_FRAGMENT_PROGRAM_ARB, 0, width, 0, 0, 0);
#else
    // download image from PBO to OpenGL texture
    glBindBufferARB(GL_PIXEL_UNPACK_BUFFER_ARB, pbo);
    glBindTexture(GL_TEXTURE_RECTANGLE_ARB, displayTex);
    glPixelStorei(GL_UNPACK_ALIGNMENT, 1);
    glTexSubImage2D(GL_TEXTURE_RECTANGLE_ARB, 0, 0, 0, width, height, GL_BGRA, GL_UNSIGNED_BYTE, 0);
    glEnable(GL_TEXTURE_RECTANGLE_ARB);
#endif

    // draw textured quad
    glDisable(GL_DEPTH_TEST);
    glBegin(GL_QUADS);
    glTexCoord2f(0, height);        glVertex2f(0, 0);
    glTexCoord2f(width, height);    glVertex2f(1, 0);
    glTexCoord2f(width, 0);         glVertex2f(1, 1);
    glTexCoord2f(0, 0);             glVertex2f(0, 1);
    glEnd();
    glDisable(GL_TEXTURE_RECTANGLE_ARB);
    glDisable(GL_FRAGMENT_PROGRAM_ARB);

    glBindBufferARB(GL_PIXEL_UNPACK_BUFFER_ARB, 0);

    if (drawCurves) {
        // draw spline curves
        glPushMatrix();
        glScalef(0.25, 0.25, 1.0);

        glTranslatef(0.0, 2.0, 0.0);
        glColor3f(1.0, 0.0, 0.0);
        plotCurve(w3);

        glTranslatef(1.0, 0.0, 0.0);
        glColor3f(0.0, 1.0, 0.0);
        plotCurve(w2);

        glTranslatef(1.0, 0.0, 0.0);
        glColor3f(0.0, 0.0, 1.0);
        plotCurve(w1);

        glTranslatef(1.0, 0.0, 0.0);
        glColor3f(1.0, 0.0, 1.0);
        plotCurve(w0);

        glPopMatrix();
        glColor3f(1.0, 1.0, 1.0);
    }

    glutSwapBuffers();
    glutReportErrors();
}

// GLUT callback functions
void idle()
{
    glutPostRedisplay();
}

void keyboard(unsigned char key, int /*x*/, int /*y*/)
{
    switch(key) {
        case 27:
            exit(0);
            break;
        case '1':
            mode = MODE_NEAREST;
            break;
        case '2':
            mode = MODE_BILINEAR;
            break;
        case '3':
            mode = MODE_BICUBIC;
            break;
        case '4':
            mode = MODE_FAST_BICUBIC;
            break;

        case '=':
        case '+':
            scale *= 0.5f;
            break;
        case '-':
            scale *= 2.0f;
            break;
        case 'r':
            scale = 1.0f;
            tx = ty = 0.0f;
            break;
        case 'd':
            printf("%f, %f, %f\n", tx, ty, scale);
        case 'b':
            runBenchmark(500);
            break;
        case 'c':
            drawCurves ^= 1;
            break;
        default:
            break;
    }

    glutPostRedisplay();
}

int ox, oy;
int buttonState = 0;

void mouse(int button, int state, int x, int y)
{
    if (state == GLUT_DOWN)
        buttonState |= 1<<button;
    else if (state == GLUT_UP)
        buttonState = 0;

    ox = x; oy = y;
    glutPostRedisplay();
}

void motion(int x, int y)
{
    float dx, dy;
    dx = x - ox;
    dy = y - oy;

    if (buttonState & 1) {
        // left = translate
        tx -= dx*scale;
        ty -= dy*scale;
    }
    else if (buttonState & 2) {
        // middle = zoom
        scale -= dy / 1000.0;
    }

    ox = x; oy = y;
    glutPostRedisplay();
}

void reshape(int x, int y)
{
    width = x; height = y;

    initPixelBuffer();

    glViewport(0, 0, x, y);

    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();

    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    glOrtho(0.0, 1.0, 0.0, 1.0, 0.0, 1.0); 
}

void cleanup()
{
    cutilSafeCall(hipFreeArray(d_imageArray));
	cutilSafeCall(cudaGLUnregisterBufferObject(pbo));    
	glDeleteBuffersARB(1, &pbo);
#if USE_BUFFER_TEX
    glDeleteTextures(1, &bufferTex);
#else
    glDeleteTextures(1, &displayTex);
#endif
    glDeleteProgramsARB(1, &fprog);
}

int iDivUp(int a, int b){
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

void initPixelBuffer()
{
    if (pbo) {
        // delete old buffer
        cutilSafeCall(cudaGLUnregisterBufferObject(pbo));
        glDeleteBuffersARB(1, &pbo);
    }

    // create pixel buffer object for display
    glGenBuffersARB(1, &pbo);
	glBindBufferARB(GL_PIXEL_UNPACK_BUFFER_ARB, pbo);
	glBufferDataARB(GL_PIXEL_UNPACK_BUFFER_ARB, width*height*sizeof(uchar4), 0, GL_STREAM_DRAW_ARB);
	glBindBufferARB(GL_PIXEL_UNPACK_BUFFER_ARB, 0);

	cutilSafeCall(cudaGLRegisterBufferObject(pbo));

#if USE_BUFFER_TEX
    // create buffer texture, attach to pbo
    if (bufferTex) {
        glDeleteTextures(1, &bufferTex);
    }
    glGenTextures(1, &bufferTex);
    glBindTexture(GL_TEXTURE_BUFFER_EXT, bufferTex);
    glTexBufferEXT(GL_TEXTURE_BUFFER_EXT, GL_RGBA8, pbo);
    glBindTexture(GL_TEXTURE_BUFFER_EXT, 0);
#else
    // create texture for display
    if (displayTex) {
        glDeleteTextures(1, &displayTex);
    }
    glGenTextures(1, &displayTex);
    glBindTexture(GL_TEXTURE_RECTANGLE_ARB, displayTex);
    glTexImage2D(GL_TEXTURE_RECTANGLE_ARB, 0, GL_RGBA8, width, height, 0, GL_RGBA, GL_UNSIGNED_BYTE, NULL);
    glTexParameteri(GL_TEXTURE_RECTANGLE_ARB, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_RECTANGLE_ARB, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
    glBindTexture(GL_TEXTURE_RECTANGLE_ARB, 0);
#endif

    // calculate new grid size
    gridSize = dim3(iDivUp(width, blockSize.x), iDivUp(height, blockSize.y));
}

void mainMenu(int i)
{
    keyboard(i, 0, 0);
}

void initMenus()
{
    glutCreateMenu(mainMenu);
    glutAddMenuEntry("Nearest filtering [1]", '1');
    glutAddMenuEntry("Bilinear filtering [2]", '2');
    glutAddMenuEntry("Bicubic filtering [3]", '3');
    glutAddMenuEntry("Fast Bicubic filtering [4]", '4');
    glutAddMenuEntry("Zoom in [=]", '=');
    glutAddMenuEntry("Zoom out [-]", '-');
    glutAddMenuEntry("Benchmark [b]", 'b');
    glutAddMenuEntry("Quit [esc]", 27);
    glutAttachMenu(GLUT_RIGHT_BUTTON);
}

void runBenchmark(int iterations)
{
    unsigned int timer;
    cutilCheckError(cutCreateTimer(&timer));

    uchar4 *d_output;
    cutilSafeCall(cudaGLMapBufferObject((void**)&d_output, pbo));

    cutilCheckError(cutStartTimer(timer));  
    for (int i = 0; i < iterations; ++i)
    {
        render(d_output);
    }

    hipDeviceSynchronize();
    cutilCheckError(cutStopTimer(timer));  
    float time = cutGetTimerValue(timer) / (float) iterations;

    cutilSafeCall(cudaGLUnmapBufferObject(pbo));

    printf("time: %0.3f ms, %f Mpixels/sec\n", time, (width*height / (time * 0.001f)) / 1e6);    
}

// fragment program for reading from buffer texture
static const char *shaderCode = 
"!!NVfp4.0\n"
"INT PARAM width = program.local[0];\n"
"INT TEMP index;\n"
"FLR.S index, fragment.texcoord;\n"
"MAD.S index.x, index.y, width, index.x;\n" // compute 1D index from 2D coords
"TXF result.color, index.x, texture[0], BUFFER;\n"
"END";

GLuint compileASMShader(GLenum program_type, const char *code)
{
    GLuint program_id;
    glGenProgramsARB(1, &program_id);
    glBindProgramARB(program_type, program_id);
    glProgramStringARB(program_type, GL_PROGRAM_FORMAT_ASCII_ARB, (GLsizei) strlen(code), (GLubyte *) code);

    GLint error_pos;
    glGetIntegerv(GL_PROGRAM_ERROR_POSITION_ARB, &error_pos);
    if (error_pos != -1) {
        const GLubyte *error_string;
        error_string = glGetString(GL_PROGRAM_ERROR_STRING_ARB);
        fprintf(stderr, "Program error at position: %d\n%s\n", (int)error_pos, error_string);
        return 0;
    }
    return program_id;
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main( int argc, char** argv) 
{
    if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") )
		cutilDeviceInit(argc, argv);
	else
		hipSetDevice( cutGetMaxGflopsDeviceId() );

    // parse arguments
    char *filename;
    bool benchmark;

    if (argc > 1) {
        if (cutGetCmdLineArgumentstr( argc, (const char**) argv, "file", &filename)) {
            imageFilename = filename;
        }
        benchmark = cutCheckCmdLineFlag(argc, (const char**) argv, "benchmark") != 0;

        if (cutCheckCmdLineFlag(argc, (const char **)argv, "qatest")) {
            benchmark = true;
        }
        if (cutCheckCmdLineFlag(argc, (const char **)argv, "noprompt")) {
            benchmark = true;
        }
    }

    // load image from disk
    uchar* h_data = NULL;
    char* imagePath = cutFindFilePath(imageFilename, argv[0]);
    if (imagePath == 0)
        exit(EXIT_FAILURE);
    cutilCheckError(cutLoadPGMub(imagePath, &h_data, &imageWidth, &imageHeight));

    printf("Loaded '%s', %d x %d pixels\n", imageFilename, imageWidth, imageHeight);

    cx = imageWidth * 0.5f;
    cy = imageHeight * 0.5f;

    // allocate array and copy image data
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(8, 0, 0, 0, hipChannelFormatKindUnsigned);
    cutilSafeCall( hipMallocArray(&d_imageArray, &channelDesc, imageWidth, imageHeight) ); 
    uint size = imageWidth * imageHeight * sizeof(uchar);
    cutilSafeCall( hipMemcpyToArray(d_imageArray, 0, 0, h_data, size, hipMemcpyHostToDevice) );
    cutFree(h_data);

    // set texture parameters
    tex.addressMode[0] = hipAddressModeClamp;
    tex.addressMode[1] = hipAddressModeClamp;
    tex.filterMode = hipFilterModeLinear;
    tex.normalized = false;    // access with integer texture coordinates

    // Bind the array to the texture
    cutilSafeCall( hipBindTextureToArray(tex, d_imageArray, channelDesc) );

    printf(
        "Press '=' and '-' to zoom\n"
        "Press number keys to change filtering mode:\n"
        "1 - nearest filtering\n"
        "2 - bilinear filtering\n"
        "3 - bicubic filtering\n"
        "4 - fast bicubic filtering\n"
        );


    // initialize GLUT callback functions
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_RGB | GLUT_DOUBLE);
    glutInitWindowSize(width, height);
    glutCreateWindow("CUDA bicubic texture filtering");
    glutDisplayFunc(display);
    glutKeyboardFunc(keyboard);
    glutMouseFunc(mouse);
    glutMotionFunc(motion);
    glutReshapeFunc(reshape);
    glutIdleFunc(idle);

    initMenus();

    atexit(cleanup);

    glewInit();
    if (!glewIsSupported("GL_VERSION_2_0 "
                         "GL_ARB_pixel_buffer_object "
                         "GL_EXT_texture_buffer_object "
                         "GL_NV_gpu_program4 "
                         )) {
        fprintf(stderr, "Required OpenGL extensions missing.");
        exit(-1);
    }
    initPixelBuffer();
    fprog = compileASMShader(GL_FRAGMENT_PROGRAM_ARB, shaderCode);
    if (!fprog) exit(0);

    if (benchmark) {
        runBenchmark(500);
//        hipDeviceReset();
        printf("Test PASSED!\n");
        exit(0);
    }

    glutMainLoop();

    hipDeviceReset();
    return 0;
}
