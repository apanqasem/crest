#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2006 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and 
 * international Copyright laws.  
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE 
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH 
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF 
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.   
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL, 
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS 
 * OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE 
 * OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE 
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.  This source code is a "commercial item" as 
 * that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of 
 * "commercial computer software" and "commercial computer software 
 * documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995) 
 * and is provided to the U.S. Government only as a commercial end item.  
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through 
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */
 
 /*
	This file contains simple wrapper functions that call the CUDA kernels
 */

#include <cutil_inline.h>
#include <cstdlib>
#include <cstdio>
#include <string.h>
#include <GL/glew.h>
#include <cuda_gl_interop.h>

#include "particles_kernel.cu"
#include "ParticleSystem.cuh"

extern "C"
{

hipArray *noiseArray;

void initCuda()
{
    hipSetDevice( cutGetMaxGflopsDeviceId() );   
}

void setParameters(SimParams *hostParams)
{
    // copy parameters to constant memory
    cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(params), hostParams, sizeof(SimParams)) );
}

//Round a / b to nearest higher integer value
int iDivUp(int a, int b){
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

// compute grid and thread block size for a given number of elements
void computeGridSize(int n, int blockSize, int &numBlocks, int &numThreads)
{
    numThreads = min(blockSize, n);
    numBlocks = iDivUp(n, numThreads);
}

inline float frand()
{
    return rand() / (float) RAND_MAX;
}

// create 3D texture containing random values
void createNoiseTexture(int w, int h, int d)
{
    hipExtent size = make_hipExtent(w, h, d);
    uint elements = (uint) size.width*size.height*size.depth;

    float *volumeData = new float [elements*4];
    float *ptr = volumeData;
    for(uint i=0; i<elements; i++) {
        *ptr++ = frand()*2.0f-1.0f;
        *ptr++ = frand()*2.0f-1.0f;
        *ptr++ = frand()*2.0f-1.0f;
        *ptr++ = frand()*2.0f-1.0f;
    }


    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float4>();
    cutilSafeCall( hipMalloc3DArray(&noiseArray, &channelDesc, size) );

    hipMemcpy3DParms copyParams = { 0 };
    copyParams.srcPtr   = make_hipPitchedPtr((void*)volumeData, size.width*sizeof(float4), size.width, size.height);
    copyParams.dstArray = noiseArray;
    copyParams.extent   = size;
    copyParams.kind     = hipMemcpyHostToDevice;
    cutilSafeCall( hipMemcpy3D(&copyParams) );

    // set texture parameters
    noiseTex.normalized = true;                      // access with normalized texture coordinates
    noiseTex.filterMode = hipFilterModeLinear;      // linear interpolation
    noiseTex.addressMode[0] = hipAddressModeWrap;   // wrap texture coordinates
    noiseTex.addressMode[1] = hipAddressModeWrap;
    noiseTex.addressMode[2] = hipAddressModeWrap;

    // bind array to 3D texture
    cutilSafeCall(hipBindTextureToArray(noiseTex, noiseArray, channelDesc));
}

void 
integrateSystem(float4 *oldPos, float4 *newPos,
				float4 *oldVel, float4 *newVel,
                float deltaTime,
                int numParticles)
{
    int numThreads, numBlocks;
    computeGridSize(numParticles, 256, numBlocks, numThreads);

    // execute the kernel
    integrateD<<< numBlocks, numThreads >>>(newPos, newVel,
                                            oldPos, oldVel,
                                            deltaTime,
											numParticles);
    
    // check if kernel invocation generated an error
    cutilCheckMsg("Kernel execution failed");
}


void 
calcDepth(float4*  pos, 
		  float*   keys,		// output
          uint*    indices,		// output 
          float3   sortVector,
          int      numParticles)
{
    int numThreads, numBlocks;
    computeGridSize(numParticles, 256, numBlocks, numThreads);

    // execute the kernel
    calcDepthD<<< numBlocks, numThreads >>>(pos, keys, indices, sortVector, numParticles);
    cutilCheckMsg("calcDepthD execution failed");
}

}   // extern "C"
